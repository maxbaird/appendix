
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <unistd.h>

#define NUM_ELEMENTS 1<<20
#define BLOCK_SIZE 1024

#define CUDA_ERROR_CHECK(func) { gpuAssert((func), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void reduce(volatile bool *timeout, bool *executedBlocks, int *input, int *output) {

  __shared__ unsigned int block_timeout;

  /*Calculate block ID in grid */
  unsigned long long int bid = blockIdx.x + gridDim.x *
                               (blockIdx.y + gridDim.z * blockIdx.z);

  /* Copy timeout signal from host to local block variable */
  if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
    block_timeout = *timeout;
  }

  /* Return if block was previously executed */
  if(executedBlocks[bid]){
    return;
  }

  /* Preventy any warps from proceeding until timeout is copied */
  __syncthreads();

  /* Return if block_timeout is true */
  if(block_timeout){
    return;
  }

  /* Mark block as executed */
  executedBlocks[bid] = true;

  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = input[i];

  __syncthreads();

  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    if (tid % (2 * s) == 0) {
      sdata[tid] += sdata[tid + s];
    }

    __syncthreads();
  }

  if (tid == 0){
    output[blockIdx.x] = sdata[0];
  }
}

int main(){
  size_t elems = NUM_ELEMENTS;
  size_t grid_size = (size_t)(ceill((long double)elems/(long double)BLOCK_SIZE)); 

  size_t input_size = elems * sizeof(int);
  size_t output_size = grid_size * sizeof(int);

  int *deviceInput = NULL;
  int *deviceOutput = NULL;
  int *hostInput = NULL; 
  int *hostOutput = NULL; 

  hostInput = (int *)malloc(input_size);
  hostOutput = (int *)malloc(output_size);

  if(hostInput == NULL){
    fprintf(stderr, "Failed to allocate %zu bytes for input!\n", input_size);
    exit(EXIT_FAILURE);
  }

  if(hostOutput == NULL){
    fprintf(stderr, "Failed to allocate %zu bytes for output!\n", output_size);
    exit(EXIT_FAILURE);
  }

  CUDA_ERROR_CHECK(hipMalloc((void **)&deviceInput, input_size));
  CUDA_ERROR_CHECK(hipMalloc((void **)&deviceOutput, output_size));

  size_t i = 0;

  for(i = 0; i < elems; i++){
      hostInput[i] = 1;
  }

  volatile bool *timeout = NULL;
  bool complete = false;
  bool *executedBlocks = NULL;

  hipMallocManaged((void **)&timeout, sizeof(volatile bool), hipMemAttachGlobal);
  hipMallocManaged((void **)&executedBlocks, grid_size * sizeof(bool), hipMemAttachGlobal);

  memset(executedBlocks, 0, grid_size * sizeof(bool));

  *timeout = false;
  size_t interrupt_count = 0;

  CUDA_ERROR_CHECK(hipMemcpy(deviceInput, hostInput, input_size, hipMemcpyHostToDevice));

  while(!complete){
    reduce<<<grid_size, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)>>>(timeout, executedBlocks, deviceInput, deviceOutput);
    CUDA_ERROR_CHECK(hipPeekAtLastError());

    usleep(0.001);
    *timeout = true;
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    /* Check if kernel is complete */
    size_t i = 0;
    for(i = 0; i < grid_size; i++){
      if(executedBlocks[i] == false){
       break;
      } 
    }
    interrupt_count++;

    if(i == grid_size){
      complete = true;
    }else{
      *timeout = false;
    }
  }
  fprintf(stdout, "Interrupt count: %zu\n", interrupt_count);

  CUDA_ERROR_CHECK(hipMemcpy(hostOutput, deviceOutput, output_size, hipMemcpyDeviceToHost));

  for(i = 1; i < grid_size; i++){
    hostOutput[0] += hostOutput[i];
  }

  fprintf(stdout, "Result: "); 

  if(hostOutput[0] == NUM_ELEMENTS){
    fprintf(stdout, "PASS\n");
  }else{
    fprintf(stderr, "FAIL\n");
  }

  fprintf(stdout, "Sum = %d\n", hostOutput[0]);

  free(hostInput);
  free(hostOutput);

  CUDA_ERROR_CHECK(hipFree(deviceInput));
  CUDA_ERROR_CHECK(hipFree(deviceOutput));
  CUDA_ERROR_CHECK(hipFree(executedBlocks));
  CUDA_ERROR_CHECK(hipFree((void *)timeout));

  CUDA_ERROR_CHECK(hipDeviceReset());
  return EXIT_SUCCESS;
}
